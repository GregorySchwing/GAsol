#include "hip/hip_runtime.h"
/********************************************************************

GAsol rev6

Alvaro Cortes and Lucia Fusani
GlaxoSmithKline 2017

Redistribution and use in source and binary forms, with or without modification, 
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, 
this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice, 
this list of conditions and the following disclaimer in the documentation and/or 
other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors 
may be used to endorse or promote products derived from this software without 
specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND 
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED 
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, 
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, 
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY 
OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING 
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Compilation instructions:

gcc gasol.c -o gasol -O3 -lm -fopenmp

To select the number of threads please use:

export OMP_NUM_THREADS=12 (bash)
setenv OMP_NUM_THREADS 12 (csh/tcsh)

Otherwise, OpenMP will determine the number of CPUs autmatically and adjust the number
of threads accordingly.

Rev6 - Added water numbers check 
       Fixed bug in water-water distance threshold check

Rev5 - Added ratio parameter to allow very low density waters (ratio=population/radius)
       Renamed program to "GAsol" or Genetic Algorithm for SOLvent placement

Rev4 - Fixed bugs related to ligand parsing
       Added molarity parameter

Rev3 - First version based on population pre-calculation for speed
       Added option to read a PDB file with a ligand to set centre of the sphere

Please send any feedback to alvaro.x.cortes@gsk.com


********************************************************************/

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <getopt.h>
#include <time.h>

#include "GasolDeviceMemory.h"
#include "GasolCUDAKernels.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
void show_usage()
{

        fprintf(stderr,"Usage: gasol <params>\n\n");
        fprintf(stderr,"Valid params:\n");
        fprintf(stderr,"\t-d or --dx <DX file>. Set the grid filename.\n");
        fprintf(stderr,"\t-x <value>. Set the center of the spatial filter (optional).\n");
        fprintf(stderr,"\t-y <value>. Set the center of the spatial filter (optional).\n");
        fprintf(stderr,"\t-z <value>. Set the center of the spatial filter (optional).\n");
        fprintf(stderr,"\t-r or --radius <value>. Set the radius of the spatial filter (optional).\n");
        fprintf(stderr,"\t-t or --threshold <value>. Set the minimum density threshold (default 5.0).\n");
        fprintf(stderr,"\t-p or --population <value>. Set the population for the genetic algorithm (default number of genes x 10).\n");
        fprintf(stderr,"\t-i or --iterations <value>. Set the number of generations for the genetic algorithm (default 5000).\n");
        fprintf(stderr,"\t-l or --ligand <PDB file>. Use a ligand to set the center of spatial filter (optional).\n");
        fprintf(stderr,"\t-m or --molarity <value>. Define concentration in 3D-RISM calculation (default 55.5 M).\n");
        fprintf(stderr,"\t-s or --seed <value>. Seed for the random number generator (Default -1 for current time).\n");
        fprintf(stderr,"\t-c or --ratio <value>. Minimum ratio of density/radius to consider a water site (Default: 0.15).\n");

}


/* Generates a random number between 0 and 1*/
double r2()
{
    return (double)rand() / (double)RAND_MAX ;
}

/* Random integer in an interval */
unsigned int rand_interval(unsigned int min, unsigned int max)
{
    int r;
    const unsigned int range = 1 + max - min;
    const unsigned int buckets = RAND_MAX / range;
    const unsigned int limit = buckets * range;

    do
    {
        r = rand();
    } while (r >= limit);

    return min + (r / buckets);
}

int get_COM_ligand(char *name, float *x, float *y, float *z)
{
	FILE *in = NULL;
	char *buffer_line = NULL;
	float com[3], cx = 0, cy =0 , cz = 0;
	char tmp_number[20];
	int n_atoms = 0;

	com[0] = com[1] = com[2] - 0;
	if( (in = fopen(name,"rb")) == NULL)
	{
		fprintf(stderr,"Cannot open ligand file %s\n",name);
		fflush(stderr);
		return -1;
	}

        if ((buffer_line = (char *) calloc(sizeof(char),1024)) == NULL)
        {
                fprintf(stderr,"Error allocating memory\n");
                fflush(stderr);
                fclose(in);
                return -2;
        }


        while( (fgets(buffer_line,1023,in)) != NULL)
        {

		if( (buffer_line[0] == 'A' && buffer_line[1] == 'T' && buffer_line[2] == 'O' && buffer_line[3] == 'M') || ( buffer_line[0] == 'H'  && buffer_line[1] == 'E' && buffer_line[2] == 'T' && buffer_line[3] == 'A' && buffer_line[4] == 'T' && buffer_line[5] == 'M' ))
		{
			cx = cy = cz = 0.0f;
			strncpy(tmp_number,&buffer_line[30],8);
			tmp_number[8] = 0;
			cx = atof(tmp_number);
        	        strncpy(tmp_number,&buffer_line[38],8);
                        tmp_number[8] = 0;
                	cy = atof(tmp_number);
                	strncpy(tmp_number,&buffer_line[46],8);
                        tmp_number[8] = 0;
                	cz = atof(tmp_number);
			com[0] += cx;
			com[1] += cy;
			com[2] += cz;
			n_atoms++;
		}


	}

	if( n_atoms == 0)
	{
		fprintf(stderr,"PDB file contains no atoms\n");
		fflush(stderr);
		return -1;
	}

	com[0] /= (float) n_atoms;
	com[1] /= (float) n_atoms;
	com[2] /= (float) n_atoms;
	*x = com[0];
	*y = com[1];
	*z = com[2];

	free(buffer_line);
	fclose(in);

	return 0;
}


int main( int argc, char *argv[])
{

	FILE *dx_in = NULL;
	char *buffer_line = NULL;
	int lines = 0, nx = 0, ny = 0, nz = 0, i = 0, j = 0, k = 0, l =0, current_delta = 0, points = 0;
	float min[3], delta[3];
	char *token = NULL;
	int current_token = 0, state_read = 0, current_point = 0;
	float *data = NULL, *g = NULL, *max_g = NULL;
	char **line_tokens = NULL;
	int *x_index = NULL, *y_index = NULL, *z_index = NULL, n_points = 0;
	int **population = NULL, **offspring = NULL, *best_ever = NULL;
	double *fitness = NULL, *new_fitness = NULL, current_g = 0;
	int max_ind = 0, nind = 0, ngen = 0, max_ngen = 0, ig = 0, jg = 0, kg = 0, lg = 0;
        double best_fitness = -9999.0f, w_sum = 0, all_g = 0, full_integ = 0.0;
	float *points_radii = NULL;
	int **forbid_combination = NULL;
	float current_distance = 0.0f, current_population = 0.0f;
        double w[6] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
	double p1 = 0, p2 = 0, p3 = 0, p4 = 0, p5 = 0, p6 = 0;
	double d1 = 0, d2 = 0, d3 = 0, d4 = 0, d5 = 0, d6 = 0;
	double penalty = 0, e = 0;
	float dx = 0.0f, dy = 0.0f, dz = 0.0f, dr = 0.0f;
	float concentration = 55.0;

	int t1 = -1, t2 = -1, t3 = -1, best_t = -1, bad_w = 0;
	int c1 = 0, c2 = 0, cxpoint1 = 0, cxpoint2 = 0;

	float x = 0.0f, y = 0.0f, z =0.0f;
	char c;

	char *grid_name = NULL, *ligand_name = NULL;
	float select_x = 0, select_y = 0, select_z = 0, select_radius = 0, threshold = 5.0;
	float min_ratio = 0.15;
	int seed = -1;

        max_ngen = 10000;


         while (1)
         {
           static struct option long_options[] =
             {
               {"dx",   required_argument, 0, 'd'},
               {"x",   required_argument, 0, 'x'},
               {"y",  required_argument, 0, 'y'},
               {"z",  required_argument, 0, 'z'},
               {"radius",  required_argument, 0, 'r'},
               {"threshold",    required_argument, 0, 't'},
               {"population",    required_argument, 0, 'p'},
               {"iterations",    required_argument, 0, 'i'},
               {"ligand",    required_argument, 0, 'l'},
               {"molarity",    required_argument, 0, 'm'},
               {"seed",    required_argument, 0, 's'},
               {"ratio",    required_argument, 0, 'c'},
               {0, 0, 0, 0}
             };
           int option_index = 0;

           c = getopt_long (argc, argv, "d:x:y:z:t:r:p:i:l:m:s:c:",
                            long_options, &option_index);
           if (c == -1)
             break;

           switch (c)
             {
             case 0:
               /* If this option set a flag, do nothing else now. */
               if (long_options[option_index].flag != 0)
                 break;
               printf ("option %s", long_options[option_index].name);
               if (optarg)
                 printf (" with arg %s", optarg);
               printf ("\n");
               break;
             case 'c':
                min_ratio = atof(optarg);
                break;
	     case 'd':
		grid_name = optarg;
		break;
	     case 'x':
                select_x = atof(optarg);
		break;
             case 'y':
                select_y = atof(optarg);
                break;
             case 'z':
                select_z = atof(optarg);
                break;
             case 'm':
                concentration = atof(optarg);
                break;
             case 'r':
                select_radius = atof(optarg);
                select_radius *= select_radius;
                break;
             case 't':
                threshold = atof(optarg);
                break;
             case 'p':
                max_ind = atoi(optarg);
		break;
             case 'i':
                max_ngen = atoi(optarg);
                break;
	     case 'l':
                ligand_name = optarg;
                break;
	     case 's':
                seed = atoi(optarg);
		break;
             case '?':
               /* getopt_long already printed an error message. */
               break;

             default:
               abort ();
             }
         }


        if( seed == -1)
                srand(time(NULL));
        else
                srand(seed);


        fprintf(stderr,"GAsol rev6 - A program to place water molecules using density grids\n");
        fprintf(stderr,"By Alvaro Cortes and Lucia Fusani. 2017 GlaxoSmithKline\n");
        fflush(stderr);

	
	if( grid_name == NULL)
	{
		fprintf(stderr,"No grid file provided\n");
		fflush(stderr);
                show_usage();
		exit(-1);
	}

        if( (dx_in = fopen(grid_name,"rb")) == NULL)
	{
		fprintf(stderr,"Cannot open the DX file %s\n",grid_name);
		fflush(stderr);
		show_usage();
		exit(-1);
	}

	
	if ((buffer_line = (char *) calloc(sizeof(char),1024)) == NULL)
	{
		fprintf(stderr,"Error allocating memory\n");
		fflush(stderr);
		fclose(dx_in);
		exit(-2);
	}
	

	if( (line_tokens = (char **) calloc(sizeof(char *), 20)) == NULL)
	{
                fprintf(stderr,"Error allocating memory\n");
                fflush(stderr);
                fclose(dx_in);
		free(buffer_line);
                exit(-2);
	}

	if( ligand_name != NULL)
	{
		if ( get_COM_ligand(ligand_name, &select_x, &select_y, &select_z) != 0)
		{
			fprintf(stderr,"Error reading PDB file\n");
			fflush(stderr);
			exit(-1);
		}
	}

	for( i = 0; i < 20; i ++)
	{
		line_tokens[i] = (char *) calloc(sizeof(char), 30);
	}

	/* This ad-hoc grid reader is not very general. It might fail with badly */
        /* formatted DX files. Need to check with different input types */
	fprintf(stderr,"Reading grid file %s ... ",grid_name);
	fflush(stderr);
        while( (fgets(buffer_line,1023,dx_in)) != NULL)
        {	
		current_token = 0;
		token = strtok(buffer_line, " ");
		strcpy(line_tokens[current_token],token);
   		while( token != NULL ) 
   		{
			current_token++;
/*      			printf( "%i %s\n", current_token-1,token );*/
      			token = strtok(NULL, " ");
			if ( token != NULL)
			strncpy(line_tokens[current_token],token,29);
   		}

	
		if( state_read != 0){

                        for(i = 0; i < current_token; i++)
                        {
                          data[current_point] = atof(line_tokens[i]);
                          ++current_point;
                        }

		if( current_point >= (nx*ny*nz))
			break;


		}else{

			if( current_token > 3 && strcmp(line_tokens[0],"object") == 0 && strcmp(line_tokens[1],"1") == 0 && strcmp(line_tokens[3],"gridpositions") == 0)
			{
				nx = atoi(line_tokens[5]);
				ny = atoi(line_tokens[6]);
				nz = atoi(line_tokens[7]);
				if( (data = (float *) calloc(sizeof(float),nx*ny*nz)) == NULL)
				{
					fprintf(stderr,"Error allocating memory\n");
					fflush(stderr);
					exit(-2);
				}
			}else if(current_token > 2 && strcmp(line_tokens[0],"origin") == 0){
				min[0] = atof(line_tokens[1]);
				min[1] = atof(line_tokens[2]);
				min[2] = atof(line_tokens[3]);
			}else if( strcmp(line_tokens[0],"delta") == 0){
				delta[current_delta] = atof(line_tokens[1+current_delta]);
				++current_delta;
		 	}else if(strcmp(line_tokens[0],"object") == 0 && strcmp(line_tokens[1],"3") == 0 && strcmp(line_tokens[3],"array") == 0){
				points = atoi(line_tokens[9]);
#ifdef DEBUG_OVERKILL
				fprintf(stderr,"%i %i\n",points,nx*ny*nz);
				fflush(stderr);
#endif
				state_read = 1;
			}

		}
        }

        fclose(dx_in);
	fprintf(stderr," done\n");
	fflush(stderr);

	if( current_point != nx*ny*nz)
	{
		fprintf(stderr,"Grid size and total number of points do not match\n");
		fflush(stderr);
		exit(-3);
	}
	fprintf(stderr,"Grid dimensions: %i,%i,%i - Total points: %i\n",nx,ny,nz,nx*ny*nz);
	fprintf(stderr,"Grid origin: %f,%f,%f - Grid spacing: %f,%f,%f\n",min[0],min[1],min[2],delta[0],delta[1],delta[2]);


	if( select_radius > 0)
	{
		fprintf(stderr,"Spatial filter is on. Solutions centered at %f,%f,%f with radius %f\n",select_x,select_y,select_z,sqrtf(select_radius));
		fflush(stderr);
	}

	/* Count the number of grid points with g(r) values above the threshold value */
	/* and optionally use the distance threshold if defined */
	l = 0;
        for( i = 0; i < nx; i++)
        {
                for(j = 0; j < ny; j++)
                {
                        for( k = 0; k < nz; k++)
                        {
				if( select_radius > 0)
				{
	 				dx = (min[0] + delta[0]*i) - select_x;
	 				dy = (min[1] + delta[1]*j) - select_y;
	 				dz = (min[2] + delta[2]*k) - select_z;
					dr = (dx*dx) + (dy*dy) + (dz*dz);
					if( data[l] >= threshold && dr <= select_radius)
					    ++n_points;
				}else{
					if( data[l] >= threshold)
						++n_points;
				}
				++l;
			}
		}
	}
	for( i = 0; i < nx*ny*nz; i++)
	{
		if( data[i] >= threshold)
		 ++n_points;
	}

	/* Select and store information for the grid points that meet the requirements */
	x_index = (int *) calloc(sizeof(int),n_points);
	y_index = (int *) calloc(sizeof(int),n_points);
	z_index = (int *) calloc(sizeof(int),n_points);
	g = (float *) calloc(sizeof(float),n_points);
	max_g = (float *) calloc(sizeof(float),n_points);
	
	current_point = 0;
	l = 0;
	all_g = 0;
	/* First pass - Select the points above the treshold and optionally use the distance
           threshold too */
	fprintf(stderr,"Selecting grid points and calculating populations ...");
	fflush(stderr);
        full_integ = 0.0;
	for( i = 0; i < nx; i++)
	{
		for(j = 0; j < ny; j++)
		{
			for( k = 0; k < nz; k++)
			{

                                if( select_radius > 0)
                                {
                                        dx = (min[0] + delta[0]*i) - select_x;
                                        dy = (min[1] + delta[1]*j) - select_y;
                                        dz = (min[2] + delta[2]*k) - select_z;
                                        dr = (dx*dx) + (dy*dy) + (dz*dz);
					if( dr <= select_radius)
						full_integ += data[l]*delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;
					if( data[l] >= threshold && dr <= select_radius) 
					{
						x_index[current_point] = i;
						y_index[current_point] = j;
						z_index[current_point] = k;
						g[current_point] = data[l] *delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;
						all_g = all_g + data[l]*delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;
						++current_point;
					}
				}else{

					full_integ += data[l]*delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;

                                        if( data[l] >= threshold)
                                        {
                                                x_index[current_point] = i;
                                                y_index[current_point] = j;
                                                z_index[current_point] = k;
                                                g[current_point] = data[l]*delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;
                                                all_g = all_g + data[l]*delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;
                                                ++current_point;
                                        }
				}
				++l;
			}
		}
	}
    points_radii = (float *) calloc(sizeof(float),current_point);
	forbid_combination = (int **) calloc(sizeof(int *), current_point);
	for( i = 0; i < current_point; i++)
	{
		forbid_combination[i] = (int *) calloc(sizeof(int),current_point);
	}
	all_g = 0;
	/* Calculate sphere with unit population for all the candidate sites */
        #pragma omp parallel for \
                    private(ig,jg,kg,l,lg,current_distance,current_population,i,j,k,dx,dy,dz,dr) \
                    shared(x_index,y_index,z_index,data,delta,nx,ny,nz,max_g,points_radii ) \
		    reduction(+:all_g) \
                    schedule(static)
        for( l = 0; l < current_point; l++)
        {
                ig = x_index[l];
                jg = y_index[l];
                kg = z_index[l];
		lg = 0;
                current_distance = 0.9;
		current_population = data[l]*delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;
                while( current_distance < 2.6 && current_population < 1.0)
                {
			current_distance += 0.1;
			current_population = data[l]*delta[0]*delta[1]*delta[2] * concentration * 6.0221415E-4;
			lg = 0;
                        for( i = 0; i < nx; i++)
                        {
                                for(j = 0; j < ny; j++)
                                {
                                        for( k = 0; k < nz; k++)
                                        {

                                        dx = (min[0] + delta[0]*i) - (min[0] + delta[0]*ig);
                                        dy = (min[1] + delta[1]*j) - (min[1] + delta[1]*jg);
                                        dz = (min[2] + delta[2]*k) - (min[2] + delta[2]*kg);
                                        dr = (dx*dx) + (dy*dy) + (dz*dz);
                                        if( dr <= (current_distance*current_distance) && !(i == ig && j == jg && k == kg))
                                        {
						current_population += data[lg]*delta[0]*delta[1]*delta[2]*concentration*6.0221415E-4;

                                        }
					++lg;
					}
                                }
                        }
                }

		max_g[l] = current_population/current_distance;
		all_g += current_population/current_distance;
		/* Bugfix - 15/08/2017. If population is not close to 1 at 2.5 A */
		/* Very low density waters filter */
                if(max_g[l] < min_ratio)
			points_radii[l] = 999.9; /* Basically eliminates this molecule from all solutions. TODO: I should remove this site instead of making it invisible */
		else
			points_radii[l] = 1.5; /*current_distance/current_population; */

#ifdef DEBUG
	fprintf(stderr,"%i point %f density %f distance\n",l,current_population,current_distance);
	fflush(stderr);
#endif

        }

        for( l = 0; l < current_point; l++)
        {
            ig = x_index[l];
            jg = y_index[l];
            kg = z_index[l];
		
			for( lg = l+1; lg < current_point; lg++)
			{
                
                dx = (min[0] + delta[0]*x_index[lg]) - (min[0] + delta[0]*ig);
                dy = (min[1] + delta[1]*y_index[lg]) - (min[1] + delta[1]*jg);
                dz = (min[2] + delta[2]*z_index[lg]) - (min[2] + delta[2]*kg);
                dr = (dx*dx) + (dy*dy) + (dz*dz);
                /*if( dr <= (points_radii[l]*points_radii[l]) || dr <= (points_radii[lg]*points_radii[lg]))*/
                if( dr <= ((points_radii[l]+points_radii[lg])*(points_radii[l]+points_radii[lg])))
                {
					forbid_combination[l][lg] = 1;
					forbid_combination[lg][l] = 1;
				}
            }
        }

		

	fprintf(stderr," done\n");
        fprintf(stderr,"Number of predicted water molecules in the site: %f\n",full_integ);
	fflush(stderr);

	fprintf(stderr,"Chromosomes of %i genes\n",current_point);
	if( max_ind == 0)
   	  max_ind = current_point * 10;

        /* Initialize random population */
        population = (int **) calloc(sizeof(int *), max_ind);
        offspring = (int **) calloc(sizeof(int *), max_ind+1);
        fitness = (double *) calloc(sizeof(double), max_ind);
        new_fitness = (double *) calloc(sizeof(double), max_ind);
        best_ever = (int *) calloc(sizeof(int), current_point);

        for( nind = 0; nind < max_ind; nind++)
        {
                fitness[nind] = -999.9;
                population[nind] = (int *) calloc(sizeof(int),current_point+1);
                offspring[nind] = (int *) calloc(sizeof(int),current_point+1);
                for( i = 0; i < current_point; i++)
                {
                    population[nind][i] = (int) rand() % 2;
                }

        }

        /* Evolve population */
        best_fitness = -9999.9;

	fprintf(stderr,"Running Genetic algorithm for %i generations with a population of %i individuals\n",max_ngen,max_ind);
	fflush(stderr);

	GasolDeviceMemory gsm;
	gsm.allocateAndCopyToDevice(fitness,x_index, y_index, z_index, g, max_g, points_radii,
								forbid_combination, population, n_points, current_point, max_ind);

	/* Generations loop */
	/* TODO: check for convergence!! */
	// Define block and grid dimensions
    int block_size = 256; // You can adjust this value based on your GPU's capabilities
    int grid_size = (max_ind + block_size - 1) / block_size;

        for( ngen = 0; ngen < max_ngen; ngen++)
        {
		evaluatePopulationKernel<<<grid_size, block_size>>>(gsm.d_fitness,max_ind, current_point,all_g,
								gsm.d_max_g, gsm.d_population, gsm.d_forbid_combination);

		/* Only parallelized for individuals */
        #pragma omp parallel for \
                    private(nind,i,j,k,current_g,d1,d2,p2,w_sum,e,penalty,p6,d6, bad_w) \
                    shared(g, forbid_combination,max_ind, fitness, lines, current_point, all_g, x_index, y_index, z_index, points_radii ) \
                    schedule(dynamic) 
                for( nind = 0 ; nind < max_ind; nind++) /* Evaluate population */
                {
                   if( fitness[nind] < -1)
                   {
                        d1 = d2 = 0.0;
						d2 = 1.0;
                        p2 = 0.0001;

						current_g = 0;
						bad_w = 0;
						for( j = 0; j < current_point; j++)
						{
							if( population[nind][j] == 1)
							{
							current_g += max_g[j]; /* Add normalized population for on bits */

							/* Check if two water molecules are overlapping in this solution */
							for( k = j+1; k < current_point; k++)
							{
								if( population[nind][k] == 1 && forbid_combination[j][k] == 1)
								{
									d2 = 0.0;
									bad_w += 1.0;
								}

							}

							}
						}

						d1 = current_g / all_g;
                        w_sum = 0;
                        e = 1.0;
						p2 = (double) bad_w / (double) current_point;
                        penalty = 1.0;
                        for ( j = 0; j < 2; j++)
                        {
                                w_sum += w[j];
                        }
                        penalty *= p2;
                        e *= powf(d1,w[0]);
                        e *= powf(d2,w[1]);
                        e = powf(e,1.0/ (double) w_sum);
                        penalty = p2-0.0001;
                        e -= penalty;
                        fitness[nind] = e;
/*                        fitness[nind] = (d1*d2)-(p2-0.0001);*/

		   			} /* Fitness */
                } /* Individuals loop */
		// wrap raw pointer with a device_ptr 
    	thrust::device_ptr<double> fitness_ptr = thrust::device_pointer_cast(gsm.d_fitness);
		thrust::device_vector<double> fitness_dev_vector(fitness_ptr, fitness_ptr + max_ind);           // from iterator range
		thrust::host_vector<double> fitness_host_vector(fitness, fitness + max_ind); 
		thrust::host_vector<double> fitness_dev_vector_h; 
		fitness_dev_vector_h.resize(max_ind);
		fitness_dev_vector_h=fitness_dev_vector;
		for (int xxx = 0; xxx < max_ind; ++xxx)
			if (fitness_dev_vector_h[xxx]!=fitness_host_vector[xxx])
				printf("%f %f\n",fitness_dev_vector_h[xxx],fitness_host_vector[xxx]);
          // from iterator range
		assert(fitness_host_vector==fitness_dev_vector);
		printf("PASS GEN %d\n",ngen);
		exit(1);
		/* Update best solution */
                for( i = 0; i < max_ind; ++i)
                {
                        if ( fitness[i] > best_fitness)
                        {
#ifdef DEBUG
                                fprintf(stderr,"New Fitness: %f\n",fitness[i]);
#endif
                                best_fitness = fitness[i];
                                for( j = 0; j < current_point; j++)
                                {
                                 best_ever[j] = population[i][j];
#ifdef DEBUG
                                 fprintf(stderr,"%i, ", best_ever[j]);
#endif
                                }
#ifdef DEBUG
                                fprintf(stderr,"\n");
				fflush(stderr);
#endif
                        }
                }
                fprintf(stderr,"Iteration %i. Best fitness so far: %f\r",ngen,best_fitness);
                fflush(stderr);



                /* Selection round with 3 inidivuals at the same time */
                for( i = 0; i < max_ind; ++i)
                {
                        t1 = rand_interval(0,max_ind-1);
                        t2 = rand_interval(0,max_ind-1);
                        t3 = rand_interval(0,max_ind-1);
                        best_t = t1;
                        if( fitness[t1] > fitness[t2])
                            best_t = t1;
                        else
                            best_t = t2;
                        if( fitness[best_t] < fitness[t3])
                            best_t = t3;

                        for( j = 0; j < current_point; j++)
                        {
                                offspring[i][j] = population[best_t][j];
                                new_fitness[i] = fitness[t3];
                        }

                }

                /* Crossover */
                for( c1 = 1; c1 < max_ind; c1 = c1 + 2)
                {
                        c2 = c1 - 1;
                        if( r2() < 0.5)
                        {
                                new_fitness[c1] = -2;
                                new_fitness[c2] = -2;
                                cxpoint1 = rand_interval(0, current_point);
                                cxpoint2 = rand_interval(0, current_point);
                                if (cxpoint2 >= cxpoint1)
                                    cxpoint2 += 1;
                                else{
                                    j = cxpoint1;
                                    cxpoint1 = cxpoint2;
                                    cxpoint2 = j;
                                }

                                for (i = cxpoint1; i < cxpoint2; i++)
                                {
                                    j = offspring[c2][i];
                                    offspring[c2][i] = offspring[c1][i];
                                    offspring[c1][i] = j;
                                }
                        }

                }

                /* Mutation */
                for( c1 = 0; c1 < max_ind; c1++)
                {
                        if( r2() < 0.2)
                        {
                                new_fitness[c1] = -2;
                                for( i = 0; i < current_point; i++)
                                {
                                        if( r2() < 0.05)
                                        {
                                                if( offspring[c1][i] == 0)
                                                    offspring[c1][i] = 1;
                                                else
                                                     offspring[c1][i] = 0;
                                        }
                                }
                        }
                }

		/* "Puberty" */
		/* Kill the parents and promote the children */
                for( c1 = 0; c1 < max_ind; c1++)
                {
                        for( i = 0; i < current_point; i++)
                        {
                              	population[c1][i] = offspring[c1][i];
                                fitness[c1] = new_fitness[c1];
                        }
                }
	}

	/* Print best solution in the form of a PDB */
	fprintf(stderr,"\nBest solution fitness: %f\n",best_fitness);
	printf("MODEL 1\n");
	printf("REMARK Fitness: %f\n",best_fitness);
        printf("REMARK Generations: %i\n",max_ngen);
        printf("REMARK Individuals: %i\n",max_ind);
        printf("REMARK Centre x,y,z: %f,%f,%f\n",select_x,select_y,select_z);
	printf("REMARK Grid: %s\n",grid_name);
	printf("REMARK Radius: %f\n",sqrtf(select_radius));
	printf("REMARK Concentration: %f\n",concentration);
	printf("REMARK Ramdom_seed: %i\n",seed);
	i = 0;
        for( j = 0; j < current_point; j++)
        {
		if( best_ever[j] == 1)
		{
			++i;
			x = min[0] + delta[0]*(x_index[j]);
			y = min[1] + delta[1]*(y_index[j]);
			z = min[2] + delta[2]*(z_index[j]);
		printf("ATOM      1  %s   HOH A%4i    %8.3f%8.3f%8.3f  1.00 %2.7f\n", "O",j,x,y,z,max_g[j]);
		}
        }
	printf("TER\n");
	printf("ENDMDL\n");
	fflush(stdout);
	if( ceil(full_integ) < i)
	{
		fprintf(stderr,"Warning: current solution has %i more water molecule/s than the integral of g(r)\n",i- (int) ceil(full_integ));
		fprintf(stderr,"It may mean that some waters are partially occupied sites or false positives\n");
		fprintf(stderr,"Please Increase the ratio threshold with --ratio\n");
		fflush(stderr);
	}
        
	/* Save the whales and free the mallocs! */
	free(buffer_line);
	for(i = 0; i < 20; i++)
		free(line_tokens[i]);
	free(line_tokens);

	for( i = 0; i < current_point; i++)
		free(forbid_combination[i]); 
	
	free(forbid_combination);
	free(x_index); free(y_index); free(z_index);
	free(g); free(max_g);

	free(fitness); free(best_ever); free(new_fitness);
	for( i = 0; i < nind; i++)
	{
		free(population[i]);
		free(offspring[i]);
	}

	free(population); free(offspring); free(points_radii);
	free(data);
	exit(0);
}
